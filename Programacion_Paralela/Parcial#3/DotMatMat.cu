
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TpB 32
#define BpG 32
__global__ void MatMulKernel ( int * MA, int * MB, int * MY, int N)
{
  __shared__ int sharedMA[TpB][TpB];
  __shared__ int sharedMB[TpB][TpB];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by * TpB + ty;
  int col = bx * TpB + tx;

  int Yval = 0;
  for (int t = 0; t<N/TpB;++t){
  	sharedMA[ty][tx] = MA[row * N + t * TpB + tx];
  	sharedMB[ty][tx] = MB[(t * TpB + ty) * N + col];
  	__syncthreads();
  	for (int k = 0; k < TpB; ++k){
  		Yval += sharedMA[ty][k] * sharedMB[k][tx];
  	}
  	__syncthreads();
  }
  MY[row * N + col] = Yval;
}
/*--( Support functions )------------------------------------------------*/

/* Initialise matrices of the given size */


/* Print matrix of the given size */
void PrintMatrix ( int *matrix, int size )
{
  int row,
      col;

  for ( row = 0; row < size; row++ )
  {
    for ( col = 0; col < size; col++ )
      printf ( "%8d ", matrix [row * size + col] );
    printf ( "\n" );
  }
  printf ( "\n" );
}

/*--( Main function )----------------------------------------------------*/

int main ( int argc, char *argv[] )
{
  int *MA,
      *MB,
      *MY;
  int *d_MA,
      *d_MB,
      *d_MY;
  int N,
      size,
      TpBx,
      TpBy;


  scanf("%d", &N);
    size = N * N * sizeof(int);
    MA = (int *) malloc ( size );
    MB = (int *) malloc ( size );
    for (int i = 0; i < N * N; i++){
        scanf("%d,", &MA[i]);
    }
    for (int i = 0; i < N * N; i++){
        scanf("%d,", &MB[i]);
    }
  /* set MY = { {0} } */
  MY = (int *) malloc ( size );
  memset ( (void *) MY, 0, size );
  //Creacion de eventos para calculo de tiempo
  hipEvent_t tstart, tstop;

  hipEventCreate(&tstart);
  hipEventCreate(&tstop);

  hipEventRecord(tstart, 0);

  hipMalloc ( &d_MA, size );
  hipMalloc ( &d_MB, size );

  hipMemcpy ( d_MA, MA, size, hipMemcpyHostToDevice );
  hipMemcpy ( d_MB, MB, size, hipMemcpyHostToDevice );

  hipMalloc ( &d_MY, size );
  hipMemcpy ( d_MY, MY, size, hipMemcpyHostToDevice );

  // Creacion del grid
  dim3 gridSize ( BpG, BpG );

  //---------------------------
  // int deviceId;
  //cudaGetDevice ( &deviceId );

 // cudaDeviceProp props;
  //cudaGetDeviceProperties ( &props, deviceId );

  //int MultiProcs = props.multiProcessorCount;
 // int warpSize = props.warpSize;

  /* Launch add () kernel on GPU */
  //printf ( "GPU vector addition of %d elements using %d blocks, %d threads per block; warp size = %d\n",
  //         N, 32 * MultiProcs, THREADS_PER_BLOCK, warpSize );
  //-------------------------------
  TpBx = N / gridSize.x;
  TpBy = N / gridSize.y;
  // Creacion de las particiones en cada bloque
  dim3 blockSize ( TpBx, TpBy );

  MatMulKernel <<< gridSize, blockSize >>> ( d_MA, d_MB, d_MY, N );

  hipMemcpy ( MY, d_MY, size, hipMemcpyDeviceToHost );

  hipEventRecord( tstop, 0);
  hipEventSynchronize( tstop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, tstart, tstop);
  printf("Tiempo de ejecucion CUDA: %3.1f ms \n", elapsedTime);
  printf ( "\n%dx%d Y Matrix is \n", N, N );
  PrintMatrix ( MY, N );

  hipFree ( d_MA );
  hipFree ( d_MB );
  hipFree ( d_MY );

  free ( MA );
  free ( MB );
  free ( MY );
}